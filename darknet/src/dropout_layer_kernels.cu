#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <cstring>

#include "dropout_layer.h"
#include "dark_cuda.h"
#include "utils.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

__global__ void drop_block_kernel(float *input, int size, float *mask, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id < size) input[id] = (mask[id]) ? 0 : (input[id] * scale);
}

void forward_dropout_layer_gpu(dropout_layer l, network_state state)
{
    if (!state.train) return;
    int iteration_num = (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    // We gradually increase the block size and the probability of dropout - during the first half of the training
    float multiplier = 1.0;
    if(iteration_num < (state.net.max_batches / 2))
        multiplier = (iteration_num / (float)(state.net.max_batches / 2));

    // dropblock
    if (l.dropblock) {
        //l.probability = 1 / keep_prob
        const int max_blocks_per_channel = 10;
        const float cur_prob = l.probability * multiplier;

        int block_width = l.dropblock_size_abs * multiplier;
        int block_height = l.dropblock_size_abs * multiplier;

        if (l.dropblock_size_rel) {
            block_width = l.dropblock_size_rel * l.w * multiplier;
            block_height = l.dropblock_size_rel * l.h * multiplier;
        }

        block_width = max_val_cmp(1, block_width);
        block_height = max_val_cmp(1, block_height);

        const float part_occupied_by_block = block_width * block_height / ((float)l.w * l.h);
        const float prob_place_block = cur_prob / (part_occupied_by_block * max_blocks_per_channel);

        memset(l.rand, 0, l.batch * l.outputs * sizeof(float));

        float count_ones = 0;

        int b, k, x, y, i;
        for (b = 0; b < l.batch; b++) {
            for (k = 0; k < l.c; k++) {
                for (i = 0; i < max_blocks_per_channel; i++) {
                    float rnd = random_float();
                    //printf(" rnd = %f \n", rnd);
                    if (rnd < prob_place_block) {
                        //count_ones += block_width  *block_height;
                        const int pre_index = k*l.w*l.h + b*l.w*l.h*l.c;
                        const int x_block = rand_int(0, l.w - block_width - 1);
                        const int y_block = rand_int(0, l.h - block_height - 1);
                        for (y = y_block; y < (y_block + block_height); y++) {
                            memset(&l.rand[x_block + y*l.w + pre_index], 1, block_width * sizeof(float));
                            //for (x = x_block; x < (x_block + block_width); x++) {
                            //    const int index = x + y*l.w + pre_index;
                            //    l.rand[index] = 1;
                            //}
                        }
                    }
                }
            }
        }

        for (i = 0; i < (l.batch*l.outputs); ++i) if (l.rand[i]) count_ones++;

        cuda_push_array(l.rand_gpu, l.rand, l.batch*l.outputs);

        l.scale = (float)(l.batch*l.outputs) / (l.batch*l.outputs - count_ones);


        //printf("\n l.scale = %f, cur_prob = %f, count_ones = %f, prob_place_block = %f, block_width = %d, block_height = %d \n",
        //    l.scale, cur_prob, count_ones, prob_place_block, block_width, block_height);

        int size = l.inputs*l.batch;

        drop_block_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.input, size, l.rand_gpu, l.scale);
        CHECK_CUDA(hipPeekAtLastError());

    }
    // dropout
    else {
        int size = l.inputs*l.batch;
        cuda_random(l.rand_gpu, size);

        /*
        int i;
        for(i = 0; i < size; ++i){
            layer.rand[i] = rand_uniform();
        }
        cuda_push_array(layer.rand_gpu, layer.rand, size);
        */

        yoloswag420blazeit360noscope << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.input, size, l.rand_gpu, l.probability, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
}

void backward_dropout_layer_gpu(dropout_layer l, network_state state)
{
    if(!state.delta) return;
    //int iteration_num = (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    int size = l.inputs*l.batch;

    // dropblock
    if (l.dropblock) {
        drop_block_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.delta, size, l.rand_gpu, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
    // dropout
    else {
        yoloswag420blazeit360noscope << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.delta, size, l.rand_gpu, l.probability, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
}
